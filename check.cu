#include "hip/hip_runtime.h"
// MIT License

// Copyright (c) 2024 CUI Xin

// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:

// The above copyright notice and this permission notice shall be included in
// all copies or substantial portions of the Software.

// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.
//

#include "hip/hip_runtime.h"
#include ""
#include "main.h"

/**
 * @brief  Error Handling
 * @param err error status code
 */
inline void checkCudaErrors(hipError_t err) {
  if (hipSuccess != err) {
    fprintf(stderr, "CUDA Runtime API error: %s.\n", hipGetErrorString(err));
    return;
  }
}

/**
 * @brief CUDA Kernel Function
 *
 * @param a
 * @param b
 * @param c
 */
__global__ void add(int *a, int *b, int *c) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  for (size_t k = 0; k < 50000; k++) {
    c[tid] = a[tid] + b[tid];
  }
}

/**
 * @brief adding token extern "C" for function runtest due to its called by
 * main.cpp
 *
 * @param a
 * @param b
 * @param c
 */
extern "C" int runtest(int *host_a, int *host_b, int *host_c) {
  int *dev_a, *dev_b, *dev_c;

  /// allocate cuda gpu memory
  checkCudaErrors(hipMalloc((void **)&dev_a, sizeof(int) * datasize));
  checkCudaErrors(hipMalloc((void **)&dev_b, sizeof(int) * datasize));
  checkCudaErrors(hipMalloc((void **)&dev_c, sizeof(int) * datasize));

  /// data blocks in host are copied to device
  checkCudaErrors(hipMemcpy(dev_a, host_a, sizeof(int) * datasize,
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(dev_b, host_b, sizeof(int) * datasize,
                             hipMemcpyHostToDevice));

  /// deal with data by GPU
  add<<<datasize / 100, 100>>>(dev_a, dev_b, dev_c);
  /// data in device write back to host
  checkCudaErrors(hipMemcpy(host_c, dev_c, sizeof(int) * datasize,
                             hipMemcpyDeviceToHost));

  /// clear gpu memory
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  return 0;
}